
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_binary_objective.hpp"

namespace LightGBM {

__global__ void BoostFromScoreKernel_1_BinaryLogloss(const label_t* cuda_labels, const data_size_t num_data, double* out_cuda_init_score) {
  __shared__ label_t shared_label[CALC_INIT_SCORE_BLOCK_SIZE_BINARY];
  const unsigned int tid = threadIdx.x;
  const unsigned int i = (blockIdx.x * blockDim.x + tid) * NUM_DATA_THREAD_ADD_CALC_INIT_SCORE_BINARY;
  shared_label[tid] = 0.0f;
  __syncthreads();
  for (unsigned int j = 0; j < NUM_DATA_THREAD_ADD_CALC_INIT_SCORE_BINARY; ++j) {
    if (i + j < num_data) {
      shared_label[tid] += cuda_labels[i + j];
    }
  }
  __syncthreads();
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < CALC_INIT_SCORE_BLOCK_SIZE_BINARY) {
      shared_label[tid] += shared_label[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) {
    atomicAdd_system(out_cuda_init_score, shared_label[0]);
  }
}

__global__ void BoostFromScoreKernel_2_BinaryLogloss(double* out_cuda_init_score, const data_size_t num_data, const double sigmoid) {
  const double suml = *out_cuda_init_score;
  const double sumw = static_cast<double>(num_data);
  const double pavg = suml / sumw;
  const double init_score = log(pavg / (1.0f - pavg)) / sigmoid;
  *out_cuda_init_score = init_score;
}

void CUDABinaryLogloss::LaunchBoostFromScoreKernel() const {
  const data_size_t num_data_per_block = CALC_INIT_SCORE_BLOCK_SIZE_BINARY * NUM_DATA_THREAD_ADD_CALC_INIT_SCORE_BINARY;
  const int num_blocks = (num_data_ + num_data_per_block - 1) / num_data_per_block;
  BoostFromScoreKernel_1_BinaryLogloss<<<num_blocks, CALC_INIT_SCORE_BLOCK_SIZE_BINARY>>>(cuda_label_, num_data_, cuda_boost_from_score_);
  if (use_nccl_) {
    CUDAVector<data_size_t> tmp_num_data(1);
    CopyFromHostToCUDADevice<data_size_t>(tmp_num_data.RawData(), &num_data_, 1, __FILE__, __LINE__);
    cudaStream_t nccl_stream;
    CUDASUCCESS_OR_FATAL(cudaStreamCreate(&nccl_stream));
    CHECK(nccl_comm_ != nullptr);
    NCCLCHECK(ncclGroupStart());
    NCCLCHECK(ncclAllReduce(cuda_boost_from_score_, cuda_boost_from_score_, 1, ncclFloat64, ncclSum, *nccl_comm_, nccl_stream));
    NCCLCHECK(ncclAllReduce(tmp_num_data.RawData(), cuda_boost_from_score_, 1, ncclFloat64, ncclSum, *nccl_comm_, nccl_stream));
    NCCLCHECK(ncclGroupEnd());
    CUDASUCCESS_OR_FATAL(cudaStreamSynchronize(nccl_stream));
    CUDASUCCESS_OR_FATAL(cudaStreamDestroy(nccl_stream));
    data_size_t global_num_data = 0;
    CopyFromCUDADeviceToHost<data_size_t>(&global_num_data, tmp_num_data.RawData(), 1, __FILE__, __LINE__);
    BoostFromScoreKernel_2_BinaryLogloss<<<1, 1>>>(cuda_boost_from_score_, global_num_data, sigmoid_);
  } else {
    SynchronizeCUDADevice(__FILE__, __LINE__);
    BoostFromScoreKernel_2_BinaryLogloss<<<1, 1>>>(cuda_boost_from_score_, num_data_, sigmoid_);
    SynchronizeCUDADevice(__FILE__, __LINE__);
  }
}

template <bool USE_LABEL_WEIGHT, bool USE_WEIGHT, bool IS_OVA>
__global__ void GetGradientsKernel_BinaryLogloss(const double* cuda_scores, const label_t* cuda_labels,
  const double* cuda_label_weights, const label_t* cuda_weights, const int ova_class_id,
  const double sigmoid, const data_size_t num_data,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    const label_t cuda_label = static_cast<int>(cuda_labels[data_index]);
    const int label = IS_OVA ? (cuda_label == ova_class_id ? 1 : -1) : (cuda_label > 0 ? 1 : -1);
    const double response = -label * sigmoid / (1.0f + exp(label * sigmoid * cuda_scores[data_index]));
    const double abs_response = fabs(response);
    if (!USE_WEIGHT) {
      if (USE_LABEL_WEIGHT) {
        const double label_weight = cuda_label_weights[label];
        cuda_out_gradients[data_index] = static_cast<score_t>(response * label_weight);
        cuda_out_hessians[data_index] = static_cast<score_t>(abs_response * (sigmoid - abs_response) * label_weight);
      } else {
        cuda_out_gradients[data_index] = static_cast<score_t>(response);
        cuda_out_hessians[data_index] = static_cast<score_t>(abs_response * (sigmoid - abs_response));
      }
    } else {
      const double sample_weight = cuda_weights[data_index];
      if (USE_LABEL_WEIGHT) {
        const double label_weight = cuda_label_weights[label];
        cuda_out_gradients[data_index] = static_cast<score_t>(response * label_weight * sample_weight);
        cuda_out_hessians[data_index] = static_cast<score_t>(abs_response * (sigmoid - abs_response) * label_weight * sample_weight);
      } else {
        cuda_out_gradients[data_index] = static_cast<score_t>(response * sample_weight);
        cuda_out_hessians[data_index] = static_cast<score_t>(abs_response * (sigmoid - abs_response) * sample_weight);
      }
    }
  }
}

#define GetGradientsKernel_BinaryLogloss_ARGS \
  scores, \
  cuda_label_, \
  cuda_label_weights_, \
  cuda_weights_, \
  ova_class_id_, \
  sigmoid_, \
  num_data_, \
  gradients, \
  hessians

void CUDABinaryLogloss::LaunchGetGradientsKernel(const double* scores, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_BINARY - 1) / GET_GRADIENTS_BLOCK_SIZE_BINARY;
  if (ova_class_id_ == -1) {
    if (cuda_label_weights_ == nullptr) {
      if (cuda_weights_ == nullptr) {
        GetGradientsKernel_BinaryLogloss<false, false, false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      } else {
        GetGradientsKernel_BinaryLogloss<false, true, false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      }
    } else {
      if (cuda_weights_ == nullptr) {
        GetGradientsKernel_BinaryLogloss<true, false, false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      } else {
        GetGradientsKernel_BinaryLogloss<true, true, false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      }
    }
  } else {
    if (cuda_label_weights_ == nullptr) {
      if (cuda_weights_ == nullptr) {
        GetGradientsKernel_BinaryLogloss<false, false, true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      } else {
        GetGradientsKernel_BinaryLogloss<false, true, true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      }
    } else {
      if (cuda_weights_ == nullptr) {
        GetGradientsKernel_BinaryLogloss<true, false, true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      } else {
        GetGradientsKernel_BinaryLogloss<true, true, true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(GetGradientsKernel_BinaryLogloss_ARGS);
      }
    }
  }
}

#undef GetGradientsKernel_BinaryLogloss_ARGS

__global__ void ConvertOutputCUDAKernel_BinaryLogloss(const double sigmoid, const data_size_t num_data, const double* input, double* output) {
  const data_size_t data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  if (data_index < num_data) {
    output[data_index] = 1.0f / (1.0f + exp(-sigmoid * input[data_index]));
  }
}

void CUDABinaryLogloss::LaunchConvertOutputCUDAKernel(const data_size_t num_data, const double* input, double* output) const {
  const int num_blocks = (num_data + GET_GRADIENTS_BLOCK_SIZE_BINARY - 1) / GET_GRADIENTS_BLOCK_SIZE_BINARY;
  ConvertOutputCUDAKernel_BinaryLogloss<<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(sigmoid_, num_data, input, output);
}

__global__ void ResetOVACUDALableKernel(
  const int ova_class_id,
  const data_size_t num_data,
  label_t* cuda_label) {
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (data_index < num_data) {
    const int int_label = static_cast<int>(cuda_label[data_index]);
    cuda_label[data_index] == (int_label == ova_class_id ? 1.0f : 0.0f);
  }
}

void CUDABinaryLogloss::LaunchResetOVACUDALableKernel() const {
  Log::Warning("before LaunchResetOVACUDALableKernel, ova_class_id = %d", ova_class_id_);
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_BINARY - 1) / GET_GRADIENTS_BLOCK_SIZE_BINARY;
  ResetOVACUDALableKernel<<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_BINARY>>>(ova_class_id_, num_data_, cuda_ova_label_);
  Log::Warning("after LaunchResetOVACUDALableKernel");
}

}  // namespace LightGBM

#endif  // USE_CUDA
