
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */


#ifdef USE_CUDA

#include "cuda_leaf_splits.hpp"
#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <bool USE_INDICES>
__global__ void CUDAInitValuesKernel1(const score_t* cuda_gradients, const score_t* cuda_hessians,
  const data_size_t num_data, const data_size_t* cuda_bagging_data_indices,
  double* cuda_sum_of_gradients, double* cuda_sum_of_hessians) {
  __shared__ double shared_mem_buffer[32];
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  double gradient = 0.0f;
  double hessian = 0.0f;
  if (data_index < num_data) {
    gradient = USE_INDICES ? cuda_gradients[cuda_bagging_data_indices[data_index]] : cuda_gradients[data_index];
    hessian = USE_INDICES ? cuda_hessians[cuda_bagging_data_indices[data_index]] : cuda_hessians[data_index];
  }
  const double block_sum_gradient = ShuffleReduceSum<double>(gradient, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double block_sum_hessian = ShuffleReduceSum<double>(hessian, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[blockIdx.x] = block_sum_gradient;
    cuda_sum_of_hessians[blockIdx.x] = block_sum_hessian;
  }
}

__global__ void ReduceGradKernel(
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  const data_size_t num_data) {
  __shared__ double shared_mem_buffer[32];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[0] = sum_of_gradients;
    cuda_sum_of_hessians[0] = sum_of_hessians;
  }
}

__global__ void CUDAInitSetValuesKernel(
  const double lambda_l1,
  const double lambda_l2,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  if (threadIdx.x == 0) {
    const double sum_of_gradients = cuda_sum_of_gradients[0];
    const double sum_of_hessians = cuda_sum_of_hessians[0];
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

__global__ void CUDAInitValuesKernel2(
  const double lambda_l1,
  const double lambda_l2,
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  __shared__ double shared_mem_buffer[32];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_hessians[0] = sum_of_hessians;
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

template <bool USE_INDICES>
__global__ void CUDAInitValuesKernel3(const int16_t* cuda_gradients_and_hessians,
  const data_size_t num_data, const data_size_t* cuda_bagging_data_indices,
  double* cuda_sum_of_gradients, double* cuda_sum_of_hessians, int64_t* cuda_sum_of_hessians_hessians,
  const score_t* grad_scale_pointer, const score_t* hess_scale_pointer) {
  const score_t grad_scale = *grad_scale_pointer;
  const score_t hess_scale = *hess_scale_pointer;
  __shared__ int64_t shared_mem_buffer[32];
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  int64_t int_gradient = 0;
  int64_t int_hessian = 0;
  if (data_index < num_data) {
    int_gradient = USE_INDICES ? cuda_gradients_and_hessians[2 * cuda_bagging_data_indices[data_index] + 1] :
      cuda_gradients_and_hessians[2 * data_index + 1];
    int_hessian = USE_INDICES ? cuda_gradients_and_hessians[2 * cuda_bagging_data_indices[data_index]] :
      cuda_gradients_and_hessians[2 * data_index];
  }
  const int64_t block_sum_gradient = ShuffleReduceSum<int64_t>(int_gradient, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const int64_t block_sum_hessian = ShuffleReduceSum<int64_t>(int_hessian, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[blockIdx.x] = block_sum_gradient * grad_scale;
    cuda_sum_of_hessians[blockIdx.x] = block_sum_hessian * hess_scale;
    cuda_sum_of_hessians_hessians[blockIdx.x] = ((block_sum_gradient << 32) | block_sum_hessian);
  }
}

__global__ void ReduceGradKernel(
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  int64_t* cuda_sum_of_gradients_hessians,
  const data_size_t num_data) {
  __shared__ double shared_mem_buffer[32];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  int64_t thread_sum_of_gradients_hessians = 0;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
    thread_sum_of_gradients_hessians += cuda_sum_of_gradients_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const int64_t sum_of_gradients_hessians = ShuffleReduceSum<int64_t>(
    thread_sum_of_gradients_hessians,
    reinterpret_cast<int64_t*>(shared_mem_buffer),
    blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[0] = sum_of_gradients;
    cuda_sum_of_hessians[0] = sum_of_hessians;
    cuda_sum_of_gradients_hessians[0] = sum_of_gradients_hessians;
  }
}

__global__ void CUDAInitSetValuesKernel(
  const double lambda_l1,
  const double lambda_l2,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  int64_t* cuda_sum_of_gradients_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  if (threadIdx.x == 0) {
    const double sum_of_gradients = cuda_sum_of_gradients[0];
    const double sum_of_hessians = cuda_sum_of_hessians[0];
    const int64_t sum_of_gradients_hessians = cuda_sum_of_gradients_hessians[0];
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->sum_of_gradients_hessians = sum_of_gradients_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

__global__ void CUDAInitValuesKernel4(
  const double lambda_l1,
  const double lambda_l2,
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  int64_t* cuda_sum_of_gradients_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  __shared__ double shared_mem_buffer[32];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  int64_t thread_sum_of_gradients_hessians = 0;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
    thread_sum_of_gradients_hessians += cuda_sum_of_gradients_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_gradients_hessians = ShuffleReduceSum<int64_t>(
    thread_sum_of_gradients_hessians,
    reinterpret_cast<int64_t*>(shared_mem_buffer),
    blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_hessians[0] = sum_of_hessians;
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->sum_of_gradients_hessians = sum_of_gradients_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

__global__ void InitValuesEmptyKernel(CUDALeafSplitsStruct* cuda_struct) {
  cuda_struct->leaf_index = -1;
  cuda_struct->sum_of_gradients = 0.0f;
  cuda_struct->sum_of_hessians = 0.0f;
  cuda_struct->num_data_in_leaf = 0;
  cuda_struct->gain = 0.0f;
  cuda_struct->leaf_value = 0.0f;
  cuda_struct->data_indices_in_leaf = nullptr;
  cuda_struct->hist_in_leaf = nullptr;
}

void CUDALeafSplits::LaunchInitValuesEmptyKernel() {
  InitValuesEmptyKernel<<<1, 1>>>(cuda_struct_);
}

void CUDALeafSplits::LaunchInitValuesKernal(
  const double lambda_l1, const double lambda_l2,
  const data_size_t* cuda_bagging_data_indices,
  const data_size_t* cuda_data_indices_in_leaf,
  const data_size_t num_used_indices,
  hist_t* cuda_hist_in_leaf) {
  if (cuda_bagging_data_indices == nullptr) {
    CUDAInitValuesKernel1<false><<<num_blocks_init_from_gradients_, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      cuda_gradients_, cuda_hessians_, num_used_indices, nullptr, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_);
  } else {
    CUDAInitValuesKernel1<true><<<num_blocks_init_from_gradients_, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      cuda_gradients_, cuda_hessians_, num_used_indices, cuda_bagging_data_indices, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_);
  }
  if (use_nccl_) {
    ReduceGradKernel<<<1, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(num_blocks_init_from_gradients_, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_, num_used_indices);
    SynchronizeCUDADevice(__FILE__, __LINE__);

    cudaStream_t cuda_stream;
    CUDASUCCESS_OR_FATAL(cudaStreamCreate(&cuda_stream));
    NCCLCHECK(ncclGroupStart());
    NCCLCHECK(ncclAllReduce(cuda_sum_of_gradients_buffer_, cuda_sum_of_gradients_buffer_, 1, ncclFloat64, ncclSum, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(cuda_sum_of_hessians_buffer_, cuda_sum_of_hessians_buffer_, 1, ncclFloat64, ncclSum, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclGroupEnd());
    CUDASUCCESS_OR_FATAL(cudaStreamSynchronize(cuda_stream));
    CUDASUCCESS_OR_FATAL(cudaStreamDestroy(cuda_stream));
    CUDAInitSetValuesKernel<<<1, 1>>>(lambda_l1, lambda_l2, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_, num_used_indices,
      cuda_data_indices_in_leaf, cuda_hist_in_leaf, cuda_struct_);
    SynchronizeCUDADevice(__FILE__, __LINE__);
  } else {
    SynchronizeCUDADevice(__FILE__, __LINE__);
    CUDAInitValuesKernel2<<<1, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      lambda_l1, lambda_l2,
      num_blocks_init_from_gradients_,
      cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_,
      num_used_indices,
      cuda_data_indices_in_leaf,
      cuda_hist_in_leaf,
      cuda_struct_);
    SynchronizeCUDADevice(__FILE__, __LINE__);
  }
}

void CUDALeafSplits::LaunchInitValuesKernal(
  const double lambda_l1, const double lambda_l2,
  const data_size_t* cuda_bagging_data_indices,
  const data_size_t* cuda_data_indices_in_leaf,
  const data_size_t num_used_indices,
  hist_t* cuda_hist_in_leaf,
  const score_t* grad_scale,
  const score_t* hess_scale) {
  if (cuda_bagging_data_indices == nullptr) {
    CUDAInitValuesKernel3<false><<<num_blocks_init_from_gradients_, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      reinterpret_cast<const int16_t*>(cuda_gradients_), num_used_indices, nullptr, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_, cuda_sum_of_gradients_hessians_buffer_, grad_scale, hess_scale);
  } else {
    CUDAInitValuesKernel3<true><<<num_blocks_init_from_gradients_, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      reinterpret_cast<const int16_t*>(cuda_gradients_), num_used_indices, cuda_bagging_data_indices, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_, cuda_sum_of_gradients_hessians_buffer_, grad_scale, hess_scale);
  }

  if (use_nccl_) {
    ReduceGradKernel<<<1, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(num_blocks_init_from_gradients_,
      cuda_sum_of_gradients_buffer_, cuda_sum_of_hessians_buffer_, cuda_sum_of_gradients_hessians_buffer_,
      num_used_indices);
    SynchronizeCUDADevice(__FILE__, __LINE__);

    cudaStream_t cuda_stream;
    CUDASUCCESS_OR_FATAL(cudaStreamCreate(&cuda_stream));
    NCCLCHECK(ncclGroupStart());
    NCCLCHECK(ncclAllReduce(cuda_sum_of_gradients_buffer_, cuda_sum_of_gradients_buffer_, 1, ncclFloat64, ncclSum, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(cuda_sum_of_hessians_buffer_, cuda_sum_of_hessians_buffer_, 1, ncclFloat64, ncclSum, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(cuda_sum_of_gradients_hessians_buffer_, cuda_sum_of_gradients_hessians_buffer_, 1, ncclInt64, ncclSum, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclGroupEnd());
    CUDASUCCESS_OR_FATAL(cudaStreamSynchronize(cuda_stream));
    CUDASUCCESS_OR_FATAL(cudaStreamDestroy(cuda_stream));
    CUDAInitSetValuesKernel<<<1, 1>>>(lambda_l1, lambda_l2, cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_, cuda_sum_of_gradients_hessians_buffer_, num_used_indices,
      cuda_data_indices_in_leaf, cuda_hist_in_leaf, cuda_struct_);
    SynchronizeCUDADevice(__FILE__, __LINE__);
  } else {
    SynchronizeCUDADevice(__FILE__, __LINE__);
    CUDAInitValuesKernel4<<<1, NUM_THRADS_PER_BLOCK_LEAF_SPLITS>>>(
      lambda_l1, lambda_l2,
      num_blocks_init_from_gradients_,
      cuda_sum_of_gradients_buffer_,
      cuda_sum_of_hessians_buffer_,
      cuda_sum_of_gradients_hessians_buffer_,
      num_used_indices,
      cuda_data_indices_in_leaf,
      cuda_hist_in_leaf,
      cuda_struct_);
    SynchronizeCUDADevice(__FILE__, __LINE__);
  }
}

}  // namespace LightGBM

#endif  // USE_CUDA
